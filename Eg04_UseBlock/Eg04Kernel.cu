#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

#define DATA_SIZE 1048576
#define THREAD_NUM 256
#define BLOCK_NUM 32

int data[DATA_SIZE];

//initialization
bool InitCuda()
{
	int count{ 0 };

	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i{ 0 };
	for (int i = 0; i < count; ++i) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1)
				break;
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);
	return true;

}

//generate numbers 0-9
void GenerateNumbers(int* number, int size)
{
	for (int i = 0; i < size; ++i)
	{
		number[i] = rand() % 10;
	}
}

//calculate sum of squares
__global__ static void sumOfSquares(int* num, int* result, clock_t* time)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	int sum = 0;
	if (tid == 0)
		time[bid] = clock();

	for (int i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM)
	{
		sum += num[i] * num[i] * num[i];
	}

	result[bid * THREAD_NUM + tid] = sum;
	if (tid == 0)
		time[bid + BLOCK_NUM] = clock();
}

int main()
{
	if (!InitCuda())
		return -1;

	//generate random numbers
	GenerateNumbers(data, DATA_SIZE);

	clock_t	start = clock();
	for (int n = 0; n < 20; ++n)
	{
		//copy data to GPU memory
		int* gpuData, *result;
		clock_t* time;
		hipMalloc((void**)&gpuData, sizeof(int) * DATA_SIZE);
		hipMalloc((void**)&result, sizeof(int) * THREAD_NUM * BLOCK_NUM);
		hipMalloc((void**)&time, sizeof(clock_t) * BLOCK_NUM * 2);
		hipMemcpy(gpuData, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

		//calculate in GPU
		sumOfSquares << <BLOCK_NUM, THREAD_NUM, 0 >> > (gpuData, result, time);

		int sum[THREAD_NUM * BLOCK_NUM];
		clock_t time_use[BLOCK_NUM * 2];
		hipMemcpy(&sum, result, sizeof(int) * THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
		hipMemcpy(&time_use, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
		//free memory
		hipFree(gpuData);
		hipFree(result);
		hipFree(time);

		//calculate the sum
		int finnal_sum = 0;
		for (int i = 0; i < THREAD_NUM * BLOCK_NUM; ++i)
			finnal_sum += sum[i];

		//calculate the run time
		clock_t min_start, max_end;
		min_start = time_use[0];
		max_end = time_use[BLOCK_NUM];
		for (int i = 1; i < BLOCK_NUM; ++i)
		{
			if (min_start > time_use[i])
				min_start = time_use[i];
			if (max_end < time_use[i + BLOCK_NUM])
				max_end = time_use[i + BLOCK_NUM];
		}
		printf("GPU Sum = %d, Time = %d\n", finnal_sum, max_end - min_start);
	}
	printf("GPU Time = %d\n\n", clock() - start);

	//calculate in CPU
	int finnal_sum = 0;
	for (int i = 0; i < DATA_SIZE; ++i)
		finnal_sum += data[i] * data[i] * data[i];
	printf("CPU Sum = %d\n", finnal_sum);

	system("pause");
	return 0;
}
