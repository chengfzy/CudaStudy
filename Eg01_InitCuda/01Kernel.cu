#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

#define DATA_SIZE 1048576

int data[DATA_SIZE];

//initialization
bool InitCuda()
{
	int count{ 0 };

	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i{ 0 };
	for (int i = 0; i < count; ++i) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1)
				break;
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);
	return true;

}

//generate numbers 0-9
void GenerateNumbers(int* number, int size)
{
	for (int i = 0; i < size; ++i)
	{
		number[i] = rand() % 10;
	}
}

//calculate sum of squares
__global__ static void sumOfSquares(int* num, int* result, clock_t* time)
{
	const int tid = threadIdx.x;

	int sum = 0;
	clock_t start = clock();

	for (int i = 0; i < DATA_SIZE; ++i)
	{
		sum += num[i] * num[i] * num[i];
	}
	
	*result = sum;
	*time = clock() - start;
}

int main()
{
	if (!InitCuda())
		return -1;
	
	//generate random numbers
	GenerateNumbers(data, DATA_SIZE);

	for (int n = 0; n < 20; ++n)
	{
		//copy data to GPU memory
		int* gpuData, *result;
		clock_t* time;
		hipMalloc((void**)&gpuData, sizeof(int) * DATA_SIZE);
		hipMalloc((void**)&result, sizeof(int));
		hipMalloc((void**)&time, sizeof(clock_t));
		hipMemcpy(gpuData, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

		//calculate in GPU
		sumOfSquares << <1, 1, 0 >> > (gpuData, result, time);
		int sum;
		clock_t time_use;
		hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&time_use, time, sizeof(clock_t), hipMemcpyDeviceToHost);
		//free memory
		hipFree(gpuData);
		hipFree(result);
		hipFree(time);
		printf("GPU Sum = %d, Time = %d\n", sum, time_use);
	}

	//calculate in CPU
	int sum = 0;
	for (int i = 0; i < DATA_SIZE; ++i)
		sum += data[i] * data[i] * data[i];
	printf("CPU Sum = %d\n", sum);
	
	system("pause");
	return 0;
}
